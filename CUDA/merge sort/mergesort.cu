#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>

__global__ void MergeSort(int *nums, int *temp, int n) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = 2; i < 2 * n; i *= 2) {
        int len = i;
        if (n - tid < len) len = n - tid;
        if (tid % i == 0) {
            int *seqA = &nums[tid], lenA = i / 2, j = 0;
            int *seqB = &nums[tid + lenA], lenB = len - lenA, k = 0;
            int p = tid;
            while (j < lenA && k < lenB) {
                if (seqA[j] < seqB[k]) {
                    temp[p] = seqA[j];
                    p++;
                    j++;
                } else {
                    temp[p] = seqB[k];
                    p++;
                    k++;
                }
            }
            while (j < lenA)
                temp[p] = seqA[j];
                p++;
                j++;
            while (k < lenB)
                temp[p] = seqB[k];
                p++;
                k++;
            for (int j = tid; j < tid + len; j++)
                nums[j] = temp[j];
        }
        __syncthreads();
    }
}

int main() {
    float total_time, comp_time;
        hipEvent_t total_start, total_stop, comp_start, comp_stop;
        hipEventCreate(&total_start);
        hipEventCreate(&total_stop);
        hipEventCreate(&comp_start);
        hipEventCreate(&comp_stop);
    
    int size = 100;
    int *nums = (int*)malloc(sizeof(int) * size);
    srand(time(0));
    for (int i = 0; i < size; ++i) {
        nums[i] = rand() % 3000;
    }

    int *dNums;
    hipMalloc((void**)&dNums, sizeof(int) * size);
    int *dTemp;
    hipMalloc((void**)&dTemp, sizeof(int) * size);

    hipEventRecord(total_start);

    hipMemcpy(dNums, nums, sizeof(int) * size, hipMemcpyHostToDevice);

    dim3 threadPerBlock(10);
    dim3 blockNum((size + threadPerBlock.x - 1) / threadPerBlock.x);

    hipEventRecord(comp_start);

    MergeSort<<<blockNum, threadPerBlock>>>(dNums, dTemp, size);

    hipEventRecord(comp_stop);
    hipEventSynchronize(comp_stop);
    hipEventElapsedTime(&comp_time, comp_start, comp_stop);

    hipMemcpy(nums, dNums, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);
    hipEventElapsedTime(&total_time, total_start, total_stop);

     for (int i = 0; i < size; ++i) {
         printf("%d ", nums[i]);
     }
     printf("\n");

    free(nums);
    hipFree(dNums);
    hipFree(dTemp);
    hipEventDestroy(comp_start);
    hipEventDestroy(comp_stop);
    hipEventDestroy(total_start);
    hipEventDestroy(total_stop);

    /*
    /*
     * GPU timing
     */
    printf("Total time (ms): %f\n", total_time);
    printf("Kernel time (ms): %f\n", comp_time);
    printf("Data transfer time (ms): %f\n", total_time-comp_time);

    printf("Number of numbers: %d\n", size);
}
